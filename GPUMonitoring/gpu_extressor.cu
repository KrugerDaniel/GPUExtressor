#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <atomic>
#include <thread>
#include <chrono>
#include <nvml.h>

// Variáveis atômicas para controle de execução
std::atomic<bool> running_3d{ false };
std::atomic<bool> running_mem{ false };
std::atomic<bool> running_stressCopy{ false };

// Encapsular todas as funções exportadas em extern "C"
extern "C" {

    __declspec(dllexport) void startStress3D();
    __declspec(dllexport) void stopStress3D();

    __declspec(dllexport) void startStressMemory();
    __declspec(dllexport) void stopStressMemory();

    __declspec(dllexport) void startStressCopy();
    __declspec(dllexport) void stopStressCopy();
}

hipDeviceProp_t getCudaDeviceProp(int device) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    return prop;
}

// Kernel de estresse 3D
__global__ void stress3DKernel(float* output, int width, int height, int iterations) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = x + y * width;

    if (x < width && y < height) {
        float value = 0.0f;
        for (int i = 0; i < iterations; ++i) {
            value += sinf(x * y + i) * cosf(x - y + i);
        }
        output[idx] = value;
    }
}

// Função para estressar a GPU com operações de 3D (realiza cálculos simples em paralelo)
__global__ void kernel3D(float* d_out) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 1 << 28) {  // Aumentando o número de threads processados
        d_out[idx] = sinf((float)idx) * cosf((float)idx);  // Simples cálculo trigonométrico
    }
}

// Controle do estresse 3D
void startStress3D() {
    int device = 0;
    int targetGpuUsagePercentage = 50;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);

    const size_t elementSize = sizeof(float);
    size_t targetMem = static_cast<size_t>(freeMem * (targetGpuUsagePercentage / 100.0f));
    int N = targetMem / elementSize;
    float* d_out = nullptr;

    hipDeviceProp_t prop = getCudaDeviceProp(device);
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;

    nvmlInit();
    nvmlDevice_t nvmlDevice;
    nvmlDeviceGetHandleByIndex(device, &nvmlDevice);

    hipMalloc((void**)&d_out, N * elementSize);

    int blocksPerGrid;
    int threadsPerGrid;

    while (true) {
        // Monitorar uso atual da GPU
        hipMemGetInfo(&freeMem, &totalMem);

        // Monitorar uso atual da GPU
        nvmlUtilization_t utilization;
        nvmlDeviceGetUtilizationRates(nvmlDevice, &utilization);

        if (utilization.gpu < targetGpuUsagePercentage) {
            // GPU abaixo do alvo, aumentar carga
            size_t additionalMem = static_cast<size_t>(freeMem * ((targetGpuUsagePercentage - utilization.gpu) / 100.0f));
            size_t newMemUsage = std::min(additionalMem, freeMem); // Garantir que não exceda a memória livre

            blocksPerGrid = (freeMem / elementSize + maxThreadsPerBlock - 1) / maxThreadsPerBlock;
            blocksPerGrid = static_cast<size_t>(blocksPerGrid * ((targetGpuUsagePercentage - utilization.gpu) / 100.0f));
            threadsPerGrid = static_cast<size_t>(maxThreadsPerBlock * ((targetGpuUsagePercentage - utilization.gpu) / 100.0f));

            kernel3D<<<blocksPerGrid, threadsPerGrid>>>(d_out);
            hipDeviceSynchronize();
        }
    }

    hipFree(d_out);
    nvmlShutdown();
}

void stopStress3D() {
    running_3d = false;
}

// Kernel de estresse de memória
__global__ void stressMemoryKernel(float* d_mem, int size, int iterations) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        for (int i = 0; i < iterations; ++i) {
            d_mem[idx] = sinf(d_mem[idx]) + cosf(d_mem[idx]);
        }
    }
}

// Controle do estresse de memória
void startStressMemory() {
    size_t totalMem = getCudaDeviceProp(0).totalGlobalMem;

    float* d_a;
    float* d_b;
    hipMalloc((void**)&d_a, totalMem);
    hipMalloc((void**)&d_b, totalMem);

    while (true) {}

    hipFree(d_a);
    hipFree(d_b);
}

void stopStressMemory() {
    running_mem = false;
}

// Kernel de multiplicação de matrizes
__global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < N && col < N) {
        float value = 0.0f;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

void startStressCopy() {
    int device = 0;
    size_t totalMem = getCudaDeviceProp(device).totalGlobalMem;

    void* d_data;

    while (true) {
        // Limitação da quantidade de memória alocada, chega no máximo em 89%
        hipMalloc(&d_data, totalMem);
        hipFree(d_data);
    }
}

void stopStressCopy() {
    running_stressCopy = false;
}
